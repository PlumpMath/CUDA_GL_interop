#include "hip/hip_runtime.h"
#include "cudaGLInteropHelper.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

void CudaGLInteropHelper::RegisterBufferObject( GLuint* pbo ) {
	cudaGLRegisterBufferObject(*pbo);
}

void CudaGLInteropHelper::UnregisterBufferObject( GLuint* pbo ) {
	cudaGLUnregisterBufferObject(*pbo);
}

void CudaGLInteropHelper::SetDevice(int dev_num) {
	cudaGLSetGLDevice(dev_num);
}

void CudaGLInteropHelper::GenerateGLPBO(uchar4* dptr, int h, int w) {
	cuHelper->GeneratePBO(dptr, h, w);
}

void CudaGLInteropHelper::MapBufferObject(GLuint* pbo, int h, int w) {
	uchar4* dptr = nullptr;
	cudaGLMapBufferObject((void**)&dptr, *pbo);
	this->GenerateGLPBO(dptr, h, w);
}

void CudaGLInteropHelper::UnmapBufferObject(GLuint* pbo) {
	cudaGLUnmapBufferObject(*pbo);
}