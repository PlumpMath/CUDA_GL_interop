#include "hip/hip_runtime.h"
#include "CudaHelper.h"
#include <iostream>

__global__ void fill_buffer(uchar4 * d_dst, int w, int h) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y)
	 			 + (threadIdx.y * blockDim.x) + threadIdx.x;

	 if (threadId < w*h) {

	 	d_dst[threadId].x = 255; //magRed[i*w + j];
		d_dst[threadId].y = 255;
		d_dst[threadId].z = 255;  //magBlue[i*w + j];
		d_dst[threadId].w = 255;
	 }
}

void CUDAHelper::GeneratePBO(uchar4* dptr, int h, int w) {
	const int BLOCKDIM_X = 32;
	const int BLOCKDIM_Y = 32;

	dim3 grid(iDivUp(w, BLOCKDIM_X), iDivUp(h, BLOCKDIM_Y));
	dim3 block(BLOCKDIM_X, BLOCKDIM_Y);
	// std::cout << "before launching kernel" << std::endl;
	fill_buffer<<< grid, block >>> (dptr, h, w);
}